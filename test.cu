
#include <hip/hip_runtime.h>

__global__ void matMult(float * a, float * b, int n, float * c) {
    int BLOCK_SIZE = 512;

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float sum = 0.0f;

    int ia = n * BLOCK_SIZE * by + n * ty;
    int ib = n * BLOCK_SIZE * bx * tx;
    int k;
    for (k =0; k < n;k++)
        sum += a[ia+k ] * b[ib + k*n];
    int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;


    c[ic + n * ty +tx] = sum;
}